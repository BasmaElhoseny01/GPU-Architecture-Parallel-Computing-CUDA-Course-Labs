#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <assert.h>

#define MAX_ERR 1e-3

__global__ void vector_add(float *out, float *matrix, float *vector, int N, int M)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N)
    {
        for (int i = 0; i < M; i++)
        {
            out[tid] += vector[i] * matrix[(tid * M) + i];
        }
    }
}

int main(int argc, char *argv[])
{
    char *input_path = argv[1];
    char *output_path = argv[2];

    FILE *file = fopen(input_path, "r");

    if (!file)
    {
        printf("\n Unable to open : %s ", input_path);
        return -1;
    }

    char line[5000];

    int test_cases = atoi(fgets(line, sizeof(line), file));
    printf("test_cases: %d\n", test_cases);

    for (int k = 0; k < test_cases; k++)
    {
        int matrix_rows, matrix_columns;
        const char delimiter[] = " ";
        char *matrix_line = fgets(line, sizeof(line), file);

        matrix_rows = atoi(strtok(matrix_line, delimiter));
        matrix_columns = atoi(strtok(NULL, delimiter));
        printf("\nmatrix_rows: %d\n", matrix_rows);
        printf("matrix_columns: %d\n", matrix_columns);

        // Allocate host memory
        float *matrix = (float *)malloc(sizeof(float) * matrix_rows * matrix_columns);
        float *vector = (float *)malloc(sizeof(float) * matrix_columns);
        float *out = (float *)malloc(sizeof(float) * matrix_rows);

        // initialize out with zeros
        for (int i = 0; i < matrix_rows; i++)
        {
            out[i] = 0;
        }
        for (int i = 0; i < matrix_rows; i++)
        {
            char *matrix_line = fgets(line, sizeof(line), file);
            char *token = strtok(matrix_line, delimiter);
            for (int j = 0; j < matrix_columns; j++)
            {
                printf("%d", i * matrix_columns + j);
                matrix[(i * matrix_columns) + j] = atof(token);
                token = strtok(NULL, delimiter);
            }
        }

        // each vector element is in a new line
        for (int i = 0; i < matrix_columns; i++)
        {
            char *vector_line = fgets(line, sizeof(line), file);
            vector[i] = atof(vector_line);
        }

        printf("\nMatrix\n");
        // print the matrix and the vector
        for (int i = 0; i < matrix_rows * matrix_columns; i++)
        {
            printf("%f ", matrix[i]);
        }

        printf("\nVector\n");
        for (int i = 0; i < matrix_columns; i++)
        {
            printf("%f ", vector[i]);
        }
        printf("\n");

        float *d_matrix, *d_vector, *d_out;

        // Allocate device memory
        hipMalloc((void **)&d_matrix, sizeof(float) * matrix_rows * matrix_columns);
        hipMalloc((void **)&d_vector, sizeof(float) * matrix_columns);
        hipMalloc((void **)&d_out, sizeof(float) * matrix_rows);

        // Transfer data from host to device memory
        hipMemcpy(d_matrix, matrix, sizeof(float) * matrix_rows * matrix_columns, hipMemcpyHostToDevice);
        hipMemcpy(d_vector, vector, sizeof(float) * matrix_columns, hipMemcpyHostToDevice);
        hipMemcpy(d_out, out, sizeof(float) * matrix_rows, hipMemcpyHostToDevice);

        // Executing kernel
        vector_add<<<(matrix_rows + 255) / 256, 256>>>(d_out, d_matrix, d_vector, matrix_rows, matrix_columns);

        // Transfer data back to host memory
        hipMemcpy(out, d_out, sizeof(float) * matrix_rows, hipMemcpyDeviceToHost);

        for (int i = 0; i < matrix_rows; i++)
        {
            printf("\nout[%d] = %f", i, out[i]);
        }

        // Verification :D
        for (int i = 0; i < matrix_rows; i++)
        {
            // out[i]  result
            // lOOP on rows for the output vector
            float res = 0;
            for (int j = 0; j < matrix_columns; j++)
            {
                res += matrix[j + i * matrix_columns] * vector[j];
            }
            if (fabs(out[i] - res) >= MAX_ERR)
            {
                printf("Assertion failed: Maximum error exceeded!\n");
                printf("Computed value: %f\n", out[i]);
                printf("Reference value: %f\n", res);
                printf("Absolute error: %f\n", fabs(out[i] - res));
                printf("Check The Max Error: %f\n", MAX_ERR);
            }
            assert(fabs(out[i] - res) < MAX_ERR);
        }

        printf("\nPASSED\n");

        // Write Result to the file
        // print output to file, create if it doesn't exist and override it if it does
        FILE *output_file = fopen(output_path, "w");
        if (!output_file)
        {
            printf("\n Unable to open : %s ", output_path);
            return -1;
        }
        for (int i = 0; i < matrix_rows; i++)
        {
            fprintf(output_file, "%f\n", out[i]);
        }
        // fprintf(output_file, "\n");
        fclose(output_file);

        // Deallocate device memory
        hipFree(d_matrix);
        hipFree(d_vector);
        hipFree(d_out);

        // Deallocate host memory
        free(matrix);
        free(vector);
        free(out);
    }
    fclose(file);
    return 0;
}
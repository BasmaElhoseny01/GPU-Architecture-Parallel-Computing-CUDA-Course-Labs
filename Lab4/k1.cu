#include "hip/hip_runtime.h"
// nvcc -o out_1  ./k1.cu
// ./out_1 ./input ./output 2 ./filters/avg_9_9.txt
// nvprof out_2 ./testfile.txt ./out.txt

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <assert.h>
#include <string.h>

#define MAX_ERR 1e-6

#define DIRENT_IMPLEMENTATION
#include <dirent.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define IMAGE_CHANNELS 3

// Declare Constant Memory
// Max is 400 floating element :D 
__constant__ float filter_c[20 * 20];

// Host Functions
__host__ float *read_filter(const char *file_path,int &filter_dim)
{
    FILE *file = fopen(file_path, "r");
    if (file == NULL)
    {
        printf("Error: Unable to open file %s\n", file_path);
        exit(1);
    }


    fscanf(file, "%d", &filter_dim);
    printf("Filter size: %d\n", filter_dim);

    printf("Filter Applied:\n");
    float *filter = (float *)malloc(filter_dim * filter_dim * sizeof(float));
    for (int i = 0; i < filter_dim * filter_dim; i++)
    {
        fscanf(file, "%f", &filter[i]);
        printf("%f ", filter[i]);

        if (i % filter_dim == filter_dim - 1)
        {
            printf("\n");
        }
    }

    // Close File
    fclose(file);

    return filter;
}

// Function to get the dimensions of the first image in the directory
__host__ void get_dimensions(const char* input_dir, int* width, int* height, int* channels) {
    DIR *dir;
    struct dirent *ent;

    // Open the directory
    if ((dir = opendir(input_dir)) != NULL) {
        // Iterate over each file in the directory
        while ((ent = readdir(dir)) != NULL) {
            // Filter out directories and special entries
            if (ent->d_type == DT_REG) {
                // Concatenate directory path and filename
                char file_path[256];
                snprintf(file_path, sizeof(file_path), "%s/%s", input_dir, ent->d_name);

                // Load the image using stb_image.h
                int w, h, c;
                unsigned char* image_data = stbi_load(file_path, &w, &h, &c, 0);

                if (image_data != NULL) {
                    // Assign dimensions
                    *width = w;
                    *height = h;
                    *channels = c;

                    printf("[All]Image Size: %d x %d x %d\n", *width, *height, *channels);

                    // Free the image data
                    stbi_image_free(image_data);

                    // Close the directory and return
                    closedir(dir);
                    return;
                } else {
                    fprintf(stderr, "Error loading image: %s\n", file_path);
                }
            }
        }
        // Close the directory
        closedir(dir);
    } else {
        // Error opening directory
        perror("Unable to open directory");
    }
}

// __host__ void read_images_batch
__host__ void read_image(const char *filename, float **data, int *width, int *height, int *channels)
{
    //Read Image
    unsigned char *udata = stbi_load(filename, width, height, channels, 0);

    // Host Memory Allocation & convert data from unsigned char to float
    *data = (float *)malloc((*width) * (*height) * (*channels) * sizeof(float));

    // Normlaize the data --> 0 to 1
    for (int i = 0; i < (*width) * (*height) * (*channels); i++)
    {
        (*data)[i] = (float)udata[i] / 255.0f;
    }

    if (*data == NULL)
    {
        printf("Error loading image.\n of name %s", filename);
        exit(1);
    }
    // Free the loaded image
    stbi_image_free(udata);

    printf("Image size: %d x %d x %d\n", *width, *height,*channels);
}


__host__ void write_image(const char *folder_name, char *name, float *data, int width, int height, int channels)
{
    // Create the output file path
    std::string folder(folder_name);
    std::string path = folder + "/" + (std::string)name;
 
    printf("Writing image to %s\n", path.c_str());
    
    // Allocate memory for unsigned char data
    unsigned char *unsigned_char_data = new unsigned char[width * height * channels];

    // Convert from float to unsigned char
    for (int j = 0; j < width * height * channels; ++j)
    {
        //Clipping 
        unsigned_char_data[j] = static_cast<unsigned char>(255.0f * std::max(0.0f, std::min(1.0f, data[j]))); // Clamp values to [0, 1] range
    }

    // Write the image to disk
    if (!stbi_write_png(path.c_str(), width, height, channels, unsigned_char_data, width * channels))
    {
        printf("Failed to write image to %s\n", path.c_str());
    }
    else
    {
        printf("Sucessfully written to %s\n", path.c_str());
    }

    // Free the allocated memory
    delete[] unsigned_char_data;
}

// Device Kernels
__global__ void BatchConvolution(float *image, float *output_image, int width, int height, int batch_size, int filter_dim)
{
    int outRow = blockDim.y * blockIdx.y + threadIdx.y;
    int outCol = blockDim.x * blockIdx.x + threadIdx.x;
    // int outBatch = blockDim.z * blockIdx.z + threadIdx.z;

    // Boundary Cond
    if (outRow < height && outCol < width)
    {
        float sum = 0;
        // Looping over mask :D
        for (int filterRow = 0; filterRow < filter_dim; filterRow++)
        {
            for (int filterCol = 0; filterCol < filter_dim; filterCol++)
            {
                int inRow = outRow - filter_dim / 2 + filterRow; // outRow - FilterRaduis + filterRow
                int inCol = outCol - filter_dim / 2 + filterCol; // outCol - FilterRaduis + filterCol

                // if (batch_idx<batch_size){
                // }

                // Apply boundary conditions (ghost cells)
                inRow = max(0, min(inRow, height - 1));
                inCol = max(0, min(inCol, width - 1));


                // // Check if out of Bounday --> This is useless in case of padding
                // if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width)
                // {
                    for (int c = 0; c < 3; c++)
                    {
                        // Every Channel
                        sum += filter_c[filterRow * filter_dim + filterCol] * (float)image[(inRow * width + inCol) * IMAGE_CHANNELS + c];
                    }
                // }
            }
        }
        output_image[outRow * width + outCol] = sum;
    }

}


__host__ void verify_convolution(){
        //         // Verifcation
        //         // Perform convolution
        //         for (int i = 0; i < height; i++) {
        //             for (int j = 0; j < height; j++) {
        //                 float sum =0; // Initialize output at position (i,j) to zero
        //                 // Apply filter
        //                 for (int k = 0; k < FILTER_DIM; k++) {
        //                     for (int l = 0; l < FILTER_DIM; l++) {
        //                         int ni = i - FILTER_DIM / 2 + k;
        //                         int nj = j - FILTER_DIM / 2 + l;
        //                         for (int c = 0; c < IMAGE_CHANNELS; ++c) {
        //                             // Check boundaries
        //                             if (ni >= 0 && ni < height && nj >= 0 && nj < width) {
        //                                 sum += image[ni][nj][c] * filter[k*FILTER_DIM + l];
        //                             }
        //                         }
        //                     }
        //                 }
        //             printf("%d\n",sum);
        //             printf("%d\n",output[i][j]);
        //             assert(sum-output[i][j]<MAX_ERR);
        //             }
        //         }

        //         // Process the file here
        //         // Example: Load image using ent->d_name

}

int main(int argc, char *argv[])
{

    printf("Hello World\n");

    // Input Arguments
    char *input_dir = argv[1];
    char *output_dir = argv[2];
    int batch_size = atoi(argv[3]);
    char *filter_pth = argv[4];

    // 1. Reading Filter
    int filter_dim;
    float *filter = read_filter(filter_pth,filter_dim);

    // 2. Copy Filter to Constant Memory
    hipMemcpyToSymbol(HIP_SYMBOL(filter_c), filter, filter_dim * filter_dim * sizeof(float));

    // 3. Process Images
    // Open the input directory
    DIR *dir;
    struct dirent *ent;
    if ((dir = opendir(input_dir)) != NULL)
    {
        printf("Reading Images from Directory: %s\n", input_dir);

        // Step(1) Get Images Dimensions
        int IMAGE_WIDTH, IMAGE_HEIGHT, image_channels;
        get_dimensions(input_dir, &IMAGE_WIDTH, &IMAGE_HEIGHT, &image_channels);

        // Allocate device memory for batched input
        float *d_batched_images;
        hipMalloc((void **)&d_batched_images, sizeof(float) * IMAGE_WIDTH * IMAGE_HEIGHT * IMAGE_CHANNELS * batch_size);
        
        // Allocate memory to store filenames for each image in the batch
        char **image_filenames = (char **)malloc(batch_size * sizeof(char *));

        // Counter for Batch
        int batch_counter=0;

        // Iterate over each file in the directory
        while ((ent = readdir(dir)) != NULL)
        {
            // Filter out directories and special entries
            if (ent->d_type == DT_REG)
            {
                // Step(1) Read Image
                image_filenames[batch_counter] = (char *)malloc(256 * sizeof(char)); // Assuming maximum filename length is 256
                snprintf(image_filenames[batch_counter], 256, "%s",ent->d_name);            
                
                // Concatenate directory path and filename
                char file_path[256];
                snprintf(file_path, sizeof(file_path), "%s/%s", input_dir, ent->d_name);

                printf("Reading Image: %s\n", file_path);

                float *image_data;
                int width, height, channels;
                // 3.1 Host memory allocation & Read Image and 
                // read_image(file_path, &image_data, &width, &height, &channels);
                read_image(file_path, &image_data, &width, &height, &channels);

                // // 3.2 Device Memory Allocation for input                
                // float *d_image;
                // hipMalloc((void **)&d_image, sizeof(float) * height * width * channels);

                // 3.3 Transfer input data to device memory
                hipMemcpy(d_batched_images + batch_counter * IMAGE_WIDTH * IMAGE_HEIGHT * IMAGE_CHANNELS, image_data, sizeof(float) * height * width * IMAGE_CHANNELS, hipMemcpyHostToDevice);

                // Free host memory for image data
                free(image_data);

                // Increment Batch Counter
                batch_counter++;

                if (batch_counter == batch_size /* no more images[TODO] */){
                    printf("Batch Completed\n");
                    // Complete Batch is ready then Process it :D

                    // 3.4 Host Memory Allocation for output
                    float *output = (float *)malloc(sizeof(float) * height * width * batch_counter);
                    
                    // 3.5 Device Memory Allocation for output
                    float *d_output; // Device pointer for the 2D array
                    hipMalloc((void **)&d_output, sizeof(float) * height * width *batch_counter);

        //             // for (int i = 0; i < 2; i++)
        //             // {
        //             //     for (int j = 0; j < width; j++)
        //             //     {
        //             //         printf("%f ", image_data[i * width + j]);
        //             //     }
        //             //     printf("\n");
        //             // }

              
        //             // 3.6 Convolution
        //             // Block Size
        //             dim3 threadsPerBlock(16, 16,4);
        //             // Grid Size
        //             dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        //                         (height + threadsPerBlock.y - 1) / threadsPerBlock.y,
        //                         (batch_counter + threadsPerBlock.z - 1) / threadsPerBlock.z);
        //             // Call the kernel
        //             printf("Calling Kernel\n");
        //             // BatchConvolution<<<numBlocks, threadsPerBlock>>>(d_batched_images, d_output, width, height, batch_counter, filter_dim);

        //             // // If Error occurs in Kernel Execution Show it using hipDeviceSynchronize,hipGetLastError:D
        //             // hipDeviceSynchronize();
        //             // hipError_t error = hipGetLastError();
        //             // if (error != hipSuccess)
        //             // {
        //             //     // in red
        //             //     printf("\033[1;31m");
        //             //     printf("CUDA error: %s\n", hipGetErrorString(error));
        //             //     // reset color
        //             //     printf("\033[0m");
        //             // }
                    
                    // 3.7 Transfer output data back to host memory
                    hipMemcpy(output, d_output, sizeof(float) * height * width * batch_counter, hipMemcpyDeviceToHost);

                
                    printf("Batch Processed\n");
                    printf("batch_counter: %d\n",batch_counter);


                    // Save Batched Processed Images
                    for (int i = 0; i < batch_counter; i++)
                    {
                        printf("Saving Image %d\n",i);
                        // 3.8 Save Image
                        // Concatenate directory path and filename
                        write_image(output_dir, image_filenames[i], output + (i * height * width), width, height, 1);
                    }
        //             // // // 3.8 Save Image
        //             // // // Concatenate directory path and filename
        //             // // char out_file_path[256];
        //             // // snprintf(out_file_path, sizeof(out_file_path), "%s/%s", output_dir, ent->d_name);
        //             // // write_image(output_dir, ent->d_name, output, width, height, 1);


                   // Reset Batch Counter
                    batch_counter=0;


        //             // // 3.9 Free Host Memory
        //             // free(image_data);
                    free(output);

                    // 3.10 Free Device Memory
                    hipFree(d_batched_images);
                    hipFree(d_output);

                }
            }
        }

        // Close the directory
        closedir(dir);
    }
    else
    {
        // Failed to open directory
        perror("Failed to open Input directory");
        return EXIT_FAILURE;
    }

    // Free memory allocated for the filter in host memory
    free(filter);

    // Free memory allocated for the filter in constant memory
    hipFree(filter_c);

    // // Images as Batches
    // for(int batch_idx=0;batch_idx<batch_size;batch_idx++){
    //     // 2. Reading Image
    //     // float* image = read_image(file_path){

    //     // }

    //     // read_image(input_folder_pth);
    // }
}